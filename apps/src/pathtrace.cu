#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>

#include <stream_compaction/cpu.h>
#include <stream_compaction/naive.h>
#include <stream_compaction/efficient.h>
#include <stream_compaction/thrust.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include "timer.h"
PerformanceTimer& timer()
{
    static PerformanceTimer timer;
    return timer;
}
#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

#define DEPTH_OF_FIELD 0
#define CACHE_FIRST_BOUNCE 1
#define SORT_BY_MATERIAL 1
#define ANTIALIASING 1
#define BOUNDING_BOX 0

#define AI_DENOISE 0

#define WEIGHTS 1

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the denoised image to the OpenGL PBO directly.
__global__ void sendDenosiedImageToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x  * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y  * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z  * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}


//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        // visualize t
        /*int index = x + (y * resolution.x);
        float timeToIntersect = gBuffer[index].t * 256.0;
        pbo[index].w = 0;
        pbo[index].x = timeToIntersect;
        pbo[index].y = timeToIntersect;
        pbo[index].z = timeToIntersect;*/

        // visualize position
        /*int index = x + (y * resolution.x);
        glm::vec3 color = glm::clamp(glm::abs(gBuffer[index].position * 25.f), 0.f, 255.f);
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;*/

        // visualize normal
        int index = x + (y * resolution.x);
        glm::vec3 color = glm::clamp(glm::abs(gBuffer[index].normal * 255.f), 0.f, 255.f);

        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void denoiseIteration(int step, float c_weight, float p_weight, float n_weight, glm::ivec2 resolution, GBufferPixel* gBuffer) {

    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);

    if (x >= resolution.x && y >= resolution.y) {
        return;
    }

    //kernel
    float kernel[25] = { 1.f / 16.f, 1.f / 16.f , 1.f / 16.f , 1.f / 16.f , 1.f / 16.f,
                        1.f / 16.f, 1.f / 4.f , 1.f / 4.f , 1.f / 4.f , 1.f / 16.f,
                        1.f / 16.f, 1.f / 4.f , 3.f / 8.f , 1.f / 4.f , 1.f / 16.f,
                        1.f / 16.f, 1.f / 4.f , 1.f / 4.f , 1.f / 4.f , 1.f / 16.f,
                        1.f / 16.f, 1.f / 16.f , 1.f / 16.f , 1.f / 16.f , 1.f / 16.f };

    //offset
    glm::ivec2 offset[25] = { glm::ivec2(-2, 2) ,glm::ivec2(-1, 2), glm::ivec2(0, 2) , glm::ivec2(1, 2) ,glm::ivec2(2, 2),
                        glm::ivec2(-2, 1) ,glm::ivec2(-1, 1), glm::ivec2(0, 1) , glm::ivec2(1, 1) ,glm::ivec2(2, 1),
                         glm::ivec2(-2, 0) , glm::ivec2(-1, 0) , glm::ivec2(0, 0) , glm::ivec2(1, 0) , glm::ivec2(2, 0),
                        glm::ivec2(-2, -1) ,glm::ivec2(-1, -1), glm::ivec2(0, -1) , glm::ivec2(1, -1) ,glm::ivec2(2, -1),
                        glm::ivec2(-2, -2) ,glm::ivec2(-1, -2), glm::ivec2(0, -2) , glm::ivec2(1, -2) ,glm::ivec2(2, -2) };

    glm::vec3 sum = glm::vec3(0.f);
    glm::vec3 curr_pos = gBuffer[index].position;
    glm::vec3 curr_nor = gBuffer[index].normal;
    glm::vec3 curr_color = gBuffer[index].denoise_color;

    float cum_w = 0.f;
    float weight;
    for (int i = 0; i < 25; i++) {
        glm::ivec2 temp_cords = glm::ivec2(x, y);
        temp_cords += offset[i] * step;
        temp_cords.x = glm::clamp(temp_cords.x, 0, resolution.x - 1);
        temp_cords.y = glm::clamp(temp_cords.y, 0, resolution.y - 1);
        if (temp_cords.x < resolution.x && temp_cords.y < resolution.y) {
            int temp_index = temp_cords.x + (temp_cords.y * resolution.x);

            glm::vec3 temp_color = gBuffer[temp_index].denoise_color;
            glm::vec3 t = curr_color - temp_color;
            float dist2 = glm::dot(t, t);
            float color_weight = glm::min(glm::exp(-dist2 / c_weight), 1.f);

            glm::vec3 temp_nor = gBuffer[temp_index].normal;
            t = curr_nor - temp_nor;
            dist2 = glm::dot(t, t);
            float nor_weight = glm::min(glm::exp(-dist2 / n_weight), 1.f);

            glm::vec3 temp_pos = gBuffer[temp_index].position;
            t = curr_pos - temp_pos;
            dist2 = glm::dot(t, t);
            float pos_weight = glm::min(glm::exp(-dist2 / p_weight), 1.f);


#if WEIGHTS
            weight =color_weight;
            sum += temp_color * weight * kernel[i];
            cum_w += weight * kernel[i];
#else
            sum += temp_color * kernel[i];
            cum_w += kernel[i];
#endif
        }
    }
    gBuffer[index].updated_denoise_color = sum / cum_w;

}

__global__ void pingPongGbuffer(glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);

    if (x < resolution.x && y < resolution.y) {
        gBuffer[index].denoise_color = gBuffer[index].updated_denoise_color;
    }

}

void denoiseGbuffer(const dim3 blocksPerGrid2d, const dim3 blockSize2d, glm::ivec2 resolution, float c_weight, float p_weight, float n_weight, int logFilterSize, GBufferPixel* gBuffer) {
    int step = 1;
    for (int i = 0; i < logFilterSize; i++) {
        denoiseIteration << <blocksPerGrid2d, blockSize2d >> > (step, c_weight, p_weight, n_weight, resolution, gBuffer);
        pingPongGbuffer << <blocksPerGrid2d, blockSize2d >> > (resolution, gBuffer);
        step *= 2;
    }
}


static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
static ShadeableIntersection* dev_first_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
static glm::vec3 * dev_albedo = NULL;
static glm::vec3 * dev_denoised_output = NULL;


void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    for (int i = 0; i < scene->geoms.size(); i++)
    {
        Geom& geom = scene->geoms[i];
        hipMalloc(&geom.dev_faces, geom.faceSize * sizeof(Face));
        hipMemcpy(geom.dev_faces, (scene->allFaces[i]).data(), geom.faceSize * sizeof(Face), hipMemcpyHostToDevice);

        geom.kd.channels = scene->kdTextures[i].channels;
        geom.kd.width = scene->kdTextures[i].width;
        geom.kd.height = scene->kdTextures[i].height;
        hipMalloc(&geom.kd.image, scene->kdTextures[i].width * scene->kdTextures[i].height * scene->kdTextures[i].channels * sizeof(unsigned char));
        hipMemcpy(geom.kd.image, scene->kdTextures[i].image, scene->kdTextures[i].width * scene->kdTextures[i].height * scene->kdTextures[i].channels * sizeof(unsigned char), hipMemcpyHostToDevice);

        geom.ks.channels = scene->ksTextures[i].channels;
        geom.ks.width = scene->ksTextures[i].width;
        geom.ks.height = scene->ksTextures[i].height;
        hipMalloc(&geom.ks.image, scene->ksTextures[i].width * scene->ksTextures[i].height * scene->ksTextures[i].channels * sizeof(unsigned char));
        hipMemcpy(geom.ks.image, scene->ksTextures[i].image, scene->ksTextures[i].width * scene->ksTextures[i].height * scene->ksTextures[i].channels * sizeof(unsigned char), hipMemcpyHostToDevice);

        geom.ke.channels = scene->keTextures[i].channels;
        geom.ke.width = scene->keTextures[i].width;
        geom.ke.height = scene->keTextures[i].height;
        hipMalloc(&geom.ke.image, scene->keTextures[i].width * scene->keTextures[i].height * scene->keTextures[i].channels * sizeof(unsigned char));
        hipMemcpy(geom.ke.image, scene->keTextures[i].image, scene->keTextures[i].width * scene->keTextures[i].height * scene->keTextures[i].channels * sizeof(unsigned char), hipMemcpyHostToDevice);

        geom.bump.channels = scene->bumpTextures[i].channels;
        geom.bump.width = scene->bumpTextures[i].width;
        geom.bump.height = scene->bumpTextures[i].height;
        hipMalloc(&geom.bump.image, scene->bumpTextures[i].width * scene->bumpTextures[i].height * scene->bumpTextures[i].channels * sizeof(unsigned char));
        hipMemcpy(geom.bump.image, scene->bumpTextures[i].image, scene->bumpTextures[i].width * scene->bumpTextures[i].height * scene->bumpTextures[i].channels * sizeof(unsigned char), hipMemcpyHostToDevice);
    }

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

#if CACHE_FIRST_BOUNCE
        hipMalloc(&dev_first_intersections, pixelcount* sizeof(ShadeableIntersection));
        hipMemset(dev_first_intersections, 0, pixelcount* sizeof(ShadeableIntersection));
#endif

#if AI_DENOISE
        hipMalloc(&dev_albedo, pixelcount * sizeof(glm::vec3));
        hipMemset(dev_albedo, 0, pixelcount * sizeof(glm::vec3));
        hipMalloc(&dev_denoised_output, pixelcount * sizeof(glm::vec3));
        hipMemset(dev_denoised_output, 0, pixelcount * sizeof(glm::vec3));
#endif

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    if (hst_scene != NULL) {
        for (int i = 0; i < hst_scene->geoms.size(); i++)
        {
            Geom& geom = hst_scene->geoms[i];
            hipFree(geom.dev_faces);
            hipFree(geom.kd.image);
            hipFree(geom.ks.image);
            hipFree(geom.ke.image);
            hipFree(geom.bump.image);
        }
    }
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_gBuffer);
#if CACHE_FIRST_BOUNCE
        hipFree(dev_first_intersections);
#endif

#if AI_DENOISE
        hipFree(dev_albedo);
        hipFree(dev_denoised_output);
#endif
    checkCUDAError("pathtraceFree");
}

__global__ void generateGBuffer(
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    GBufferPixel* gBuffer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        float t = shadeableIntersections[idx].t;
        gBuffer[idx].t = t;
        gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
        gBuffer[idx].position = pathSegments[idx].ray.origin + pathSegments[idx].ray.direction * t;
    }
}


__host__ __device__ glm::vec2 ConcentricSampleDisk(const glm::vec2 &point) {
    glm::vec2 uOffset = 2.f * point - glm::vec2(1, 1);
    if (uOffset.x == 0 && uOffset.y == 0)
        return glm::vec2(0, 0);
    float theta, r;
    if (std::abs(uOffset.x) > std::abs(uOffset.y)) {
        r = uOffset.x;
        theta = 0.785398f * (uOffset.y / uOffset.x);
    }
    else {
        r = uOffset.y;
        theta = 1.570796f - 0.785398f * (uOffset.x / uOffset.y);
    }
    return r * glm::vec2(std::cos(theta), std::sin(theta));
}
/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;


	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);

		segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        float antia_x = x;
        float antia_y = y;
#if ANTIALIASING
        thrust::default_random_engine rngANTIA = makeSeededRandomEngine(iter, index, traceDepth);
        thrust::uniform_real_distribution<float> uANTIA(-0.5, 0.5);

        antia_x += uANTIA(rngANTIA);
        antia_y += uANTIA(rngANTIA);

#endif
		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)antia_x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)antia_y - (float)cam.resolution.y * 0.5f)
			);
#if DEPTH_OF_FIELD
        float lensRadius = 0.8f;
        float focalDistance = 11.0f;

        thrust::uniform_real_distribution<float> uDOF(0,1);

        if (lensRadius > 0) {
            glm::vec2 pLens = lensRadius * ConcentricSampleDisk(glm::vec2(uDOF(rng), uDOF(rng)));

            float ft = glm::abs(focalDistance / segment.ray.direction.z);
            glm::vec3 pFocus = segment.ray.origin+segment.ray.direction*ft;

            segment.ray.origin += glm::vec3(pLens.x, pLens.y, 0);
            segment.ray.direction = normalize(pFocus- segment.ray.origin);
        }
#endif
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;
        glm::vec2 uv = glm::vec2(0.0f,0.0f);

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;

		// naive parse through global geoms
        //TODO BVH
		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];
            int a = geom.faceSize;
            int b = geom.type;
			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?
            else if (geom.type == OBJ)
            {
#if BOUNDING_BOX
                if (boudingBoxIntersectionTest(geom, pathSegment.ray)) {
                    t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                else t = -1;
#endif
                //Although self-writing triangle intersect can also be used to cal t, it's slower than glm::intersectRayTriangle
                //t = objTriIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_uv, outside);
            }
			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
                uv = tmp_uv;
                int x = uv.x;
                int y = uv.y;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
            intersections[path_index].geomId = hit_geom_index;
            intersections[path_index].texcoord = uv;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
    , Geom* geoms
    , int depth
    , glm::vec3* albedo
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
#if AI_DENOISE
    if (iter == 1 && depth==1) {
        if (intersection.t > 0.0f) {
            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;
            albedo[pathSegments[idx].pixelIndex] = materialColor;
            
            Geom geom = geoms[intersection.geomId];
            if (geom.type == OBJ) {
                glm::vec3 emission(0.0f);
                if (geom.ke.channels) {
                    int coordU = (int)(intersection.texcoord.x * geom.ke.width);
                    int coordV = (int)(intersection.texcoord.y * geom.ke.height);
                    int pixelID = coordV * geom.ke.width + coordU;

                    unsigned int colR = (unsigned int)geom.ke.image[pixelID * geom.ke.channels];
                    unsigned int colG = (unsigned int)geom.ke.image[pixelID * geom.ke.channels + 1];
                    unsigned int colB = (unsigned int)geom.ke.image[pixelID * geom.ke.channels + 2];
                    emission = glm::vec3(colR / 255.f, colG / 255.f, colB / 255.f);
                }
                //if emittance load emission, else load diffuse color
                if (emission.x > FLT_EPSILON || emission.y > FLT_EPSILON || emission.z > FLT_EPSILON) {
                    albedo[pathSegments[idx].pixelIndex]= (emission * 5.0f);
                }else if (geom.kd.channels) {
                    int coordU = (int)(intersection.texcoord.x * geom.kd.width);
                    int coordV = (int)(intersection.texcoord.y * geom.kd.height);
                    int pixelID = coordV * geom.kd.width + coordU;
                    //diffuse color
                    coordU = (int)(intersection.texcoord.x * geom.kd.width);
                    coordV = (int)(intersection.texcoord.y * geom.kd.height);
                    pixelID = coordV * geom.kd.width + coordU;
                    glm::vec3 diffuseColor;
                    unsigned int colR = (unsigned int)geom.kd.image[pixelID * geom.kd.channels];
                    unsigned int colG = (unsigned int)geom.kd.image[pixelID * geom.kd.channels + 1];
                    unsigned int colB = (unsigned int)geom.kd.image[pixelID * geom.kd.channels + 2];
                    diffuseColor = glm::vec3(colR / 255.f, colG / 255.f, colB / 255.f);
                    albedo[pathSegments[idx].pixelIndex] = diffuseColor;
                }
            }else if (material.emittance > 0.0f) {
                albedo[pathSegments[idx].pixelIndex] = materialColor * material.emittance;
            }
            else if (material.hasRefractive > 0.0f) {
                albedo[pathSegments[idx].pixelIndex] = material.specular.color;
            }
        }
        else {
            albedo[pathSegments[idx].pixelIndex] = glm::vec3(0.0f);
        }
         
    }
#endif
    if (intersection.t > 0.0f) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx].color *= (materialColor * material.emittance);
        pathSegments[idx].remainingBounces = 0;
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else if (pathSegments[idx].remainingBounces == 1) {
          pathSegments[idx].color = glm::vec3(0.0);
          pathSegments[idx].remainingBounces = 0;
      }
      else {
          scatterRay(pathSegments[idx], pathSegments[idx].ray.origin+intersection.t* pathSegments[idx].ray.direction, intersection, material, rng, geoms, iter, depth);
          pathSegments[idx].remainingBounces -= 1;
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      pathSegments[idx].color = glm::vec3(0.0f);
      pathSegments[idx].remainingBounces = 0;
    }
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

__global__ void pathToBuffer(GBufferPixel* gBuffer, int nPaths, glm::vec3* image, PathSegment* iterationPaths) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        gBuffer[iterationPath.pixelIndex].denoise_color = iterationPath.color;
    }
}

__global__ void finalGatherDeniose(GBufferPixel* gBuffer, int nPaths, glm::vec3* image, PathSegment* iterationPaths) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += gBuffer[iterationPath.pixelIndex].denoise_color;
    }
}


struct sortByMaterial {
    __host__ __device__ bool operator() (const ShadeableIntersection& a, const ShadeableIntersection& b){
        return a.materialId > b.materialId;
    }
};

struct isTerminate {
    __host__ __device__ bool operator()(const PathSegment& p) {
        return p.remainingBounces > 0;
    }
};
/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter, bool denoise, int filterSize, float c_weight, float p_weight, float n_weight) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
    int num_paths_origin = num_paths;
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    
    timer().startGpuTimer();
    // Empty gbuffer
    hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));
#if CACHE_FIRST_BOUNCE && !ANTIALIASING && !DEPTH_OF_FIELD
        if (depth==0 && iter!=1) {
            thrust::copy(thrust::device, dev_first_intersections, dev_first_intersections+ num_paths_origin, dev_intersections);
#if SORT_BY_MATERIAL
                thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths_origin, dev_paths, sortByMaterial());
#endif
        }
#endif
            // clean shading chunks
     hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
     bool iterationComplete = false;
     while (!iterationComplete) {
            dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
            // tracing
            computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
                depth
                , num_paths
                , dev_paths
                , dev_geoms
                , hst_scene->geoms.size()
                , dev_intersections
                );
            checkCUDAError("trace one bounce");
            hipDeviceSynchronize();
#if CACHE_FIRST_BOUNCE && !ANTIALIASING && !DEPTH_OF_FIELD
            if (iter == 1 && depth == 0)thrust::copy(thrust::device, dev_intersections, dev_intersections + num_paths_origin, dev_first_intersections);
#endif
#if SORT_BY_MATERIAL
            thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, sortByMaterial());
#endif
            if (depth == 0) {
                generateGBuffer << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_intersections, dev_paths, dev_gBuffer);
            }

            depth++;


	// TODO:
	// --- Shading Stage ---
	// Shade path segments based on intersections and generate new rays by
  // evaluating the BSDF.
  // Start off with just a big kernel that handles all the different
  // materials you have in the scenefile.
  // TODO: compare between directly shading the path segments and shading
  // path segments that have been reshuffled to be contiguous in memory.
#if AI_DENOISE
    shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
        iter,
        num_paths,
        dev_intersections,
        dev_paths,
        dev_materials,
        dev_geoms,
        depth,
        dev_albedo
        );
#else  
    shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
    iter,
    num_paths,
    dev_intersections,
    dev_paths,
    dev_materials,
    dev_geoms,
    depth,
    NULL
  );
#endif
  //thrust::remove_if(dev_paths, dev_paths+num_paths, isTerminate());
  dev_path_end = thrust::stable_partition(thrust::device, dev_paths, dev_paths + num_paths, isTerminate());
  num_paths= dev_path_end - dev_paths;
  if(num_paths==0)iterationComplete = true; // TODO: should be based off stream compaction results.
	}
    timer().endGpuTimer();
  // Assemble this iteration and apply it to the image
    if (true) {
        pathToBuffer << <numBlocksPixels, blockSize1d >> > (dev_gBuffer, num_paths_origin, dev_image, dev_paths);
        denoiseGbuffer(blocksPerGrid2d, blockSize2d, cam.resolution, c_weight, p_weight, n_weight, filterSize, dev_gBuffer);
        finalGatherDeniose << <numBlocksPixels, blockSize1d >> > (dev_gBuffer, num_paths_origin, dev_image, dev_paths);
    }
    else {
        finalGather << <numBlocksPixels, blockSize1d >> > (num_paths_origin, dev_image, dev_paths);
    }

    ///////////////////////////////////////////////////////////////////////////
#if !AI_DENOISE
    // Send results to OpenGL buffer for rendering
    //sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
#endif
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

#if AI_DENOISE
    hipMemcpy(hst_scene->state.albedo.data(), dev_albedo,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
#endif
    checkCUDAError("pathtrace copy");
}

void sendToGPU(uchar4* pbo, int iter) {
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    hipMemcpy(dev_denoised_output, hst_scene->state.output.data(), pixelcount*sizeof(glm::vec3),hipMemcpyHostToDevice);

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    sendDenosiedImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_denoised_output);
}

void showGBuffer(uchar4* pbo) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);
}